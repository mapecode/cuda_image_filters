
#include <hip/hip_runtime.h>
#define RED_MULTIPLIER 0.2986
#define GREEN_MULTIPLIER 0.587
#define BLUE_MULTIPLIER 0.114

__global__ void apply_grayscale(unsigned char *red_channel,unsigned char *green_channel,
    unsigned char *blue_channel, const unsigned int width, const unsigned int height) {
    const unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;
    const unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row < height && col < width) {
        int index = col + row * width;
        unsigned char intensity = static_cast<unsigned char>(
            red_channel[index] * RED_MULTIPLIER + green_channel[index] * GREEN_MULTIPLIER + blue_channel[index] * BLUE_MULTIPLIER
        );

        red_channel[index] = green_channel[index] = blue_channel[index] = 
            intensity < 255 ? intensity : 255;
    }
}
