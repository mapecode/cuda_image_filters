
#include <hip/hip_runtime.h>
__global__ void apply_blue(unsigned char *red_channel,unsigned char *green_channel, 
    const unsigned int width, const unsigned int height) {
    const unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;
    const unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;

    if(row < height && col < width) {
        int index = col + row * width;
        red_channel[index] = green_channel[index] = 0;
    }
}
